#include "hip/hip_runtime.h"
#pragma once
#include <ristretto.hpp.cu>

// Note: N must be >= 1.
template <size_t N> class LookupTable {
private:
    ProjectiveNielsPoint _entries[N];

public:
    __device__ LookupTable(const RistrettoPoint& p) {
        _entries[0] = p.as_projective_niels();

        for (size_t i = 1; i < N; i++) {
            _entries[i] = (p + _entries[i - 1]).as_extended().as_projective_niels();
        }
    }

    // TODO: Eventually make this non vartime. Or not, as Sunscreen doesn't require it.
    __device__ ProjectiveNielsPoint select(i8 x) {
        ProjectiveNielsPoint ret = ProjectiveNielsPoint::IDENTITY();

        size_t idx = abs(x);

        ret = x > 0 ? _entries[idx - 1] : ret;
        ret = x < 0 ? -_entries[idx - 1] : ret;

        return ret;
    }
};